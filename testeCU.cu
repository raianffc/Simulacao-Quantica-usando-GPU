#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const double *A, const double *B, double *C, int size) {
    printf("\naqui2\n");
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        C[tid] = A[tid] + B[tid];
    }
}

int main() {
    int size = 5; // Tamanho dos vetores
    double *h_A, *h_B, *h_C; // Vetores na CPU
    double *d_A, *d_B, *d_C; // Vetores na GPU

    // Aloca memória para os vetores na CPU
    h_A = (double *)malloc(size * sizeof(double));
    h_B = (double *)malloc(size * sizeof(double));
    h_C = (double *)malloc(size * sizeof(double));

    // Inicializa os vetores na CPU
    for (int i = 0; i < size; i++) {
        h_A[i] = i;
        h_B[i] = 2 * i;
    }

    // Aloca memória para os vetores na GPU
    hipMalloc((void **)&d_A, size * sizeof(double));
    hipMalloc((void **)&d_B, size * sizeof(double));
    hipMalloc((void **)&d_C, size * sizeof(double));

    // Copia os vetores da CPU para a GPU
    hipMemcpy(d_A, h_A, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size * sizeof(double), hipMemcpyHostToDevice);

    // Define o número de threads por bloco
    int threadsPerBlock = 256;
    int numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;

    // Executa o kernel de soma dos vetores
    printf("\naqui1\n");
    vectorAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, size);
    hipDeviceSynchronize();
    printf("\naqui3\n");
    // Copia o vetor resultado da GPU para a CPU
    hipMemcpy(h_C, d_C, size * sizeof(double), hipMemcpyDeviceToHost);

    // Exibe o resultado
    printf("Vetor A: ");
    for (int i = 0; i < size; i++) {
        printf("%.1f ", h_A[i]);
    }
    printf("\nVetor B: ");
    for (int i = 0; i < size; i++) {
        printf("%.1f ", h_B[i]);
    }
    printf("\nResultado da soma: ");
    for (int i = 0; i < size; i++) {
        printf("%.1f ", h_C[i]);
    }
    printf("\n");

    // Libera a memória alocada
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipError_t hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) {
        printf("Erro durante a execução do kernel: %s\n", hipGetErrorString(hipError_t));
        return 1; // Encerra o programa com um código de erro
    }

    return 0;
}
