
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <complex.h>
#include <hiprand/hiprand_kernel.h>

#define PI 3.14159265358979323846

__global__ void generateRandom(float *randomValues, unsigned int seed) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState state;
    hiprand_init(seed, tid, 0, &state);

    randomValues[tid] = hiprand_uniform(&state);
}

int main() {
    int numThreads = 1;
    int numBlocks = 1;
    int totalThreads = numThreads * numBlocks;

    float *d_randomValues;
    float *h_randomValues = (float *)malloc(totalThreads * sizeof(float));

    hipMalloc((void **)&d_randomValues, totalThreads * sizeof(float));

    generateRandom<<<numBlocks, numThreads>>>(d_randomValues, time(0));
    
    hipMemcpy(h_randomValues, d_randomValues, totalThreads * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < totalThreads; i++) {
        printf("Random Value %d: %f\n", i, h_randomValues[i]);
    }

    hipFree(d_randomValues);
    free(h_randomValues);

    return 0;
}

