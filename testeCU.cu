#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include <string.h>
#include<stdbool.h>
#include<math.h>
#include<time.h>
#include<complex.h>
#include <fftw3.h>
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>

__global__ void calculateProbabilities(hipDoubleComplex *Y, double *Z, int q) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < q) {
        Z[idx] = hipCabs(Y[idx]) * hipCabs(Y[idx]);
    }
}

double *Prepara(double N, double x, double *r, double q) {
    int tamN = (int)log2(N);
    double q1 = 1 << (2 * tamN);
    printf("Valor ideal para q: %.0f\n", q1);

    if (q < N) {
        q = 1 << (tamN + 4);
    }
    if (*r == 0) {
        int s = x;
        int i = 1;
        while (s > 1) {
            s = (int)(s * x) % ((int)N);
            i++;
        }
        *r = i;
        printf("Ordem r não informada. Ordem r calculada: %.0f\n", *r);
    } else {
        printf("Ordem r informada: %f\n", *r);
    }

    printf("Criando Z...\n");
    double *Z = (double *)malloc(q * sizeof(double));
    if (Z == NULL) {
        printf("Erro na alocacao de memoria.");
        exit(1);
    }

    hipDoubleComplex *Y;
    hipMalloc((void **)&Y, q * sizeof(hipDoubleComplex));
    hipMemset(Y, make_hipDoubleComplex(0.0, 0.0), q * sizeof(hipDoubleComplex));

    calculateProbabilities<<<(q + 255) / 256, 256>>>(Y, Z, q);

    hipFree(Y);

    double sum_Z = 0;
    for (int i = 0; i < q; i++) {
        sum_Z += Z[i];
    }

    double temp = 0;
    for (int i = 0; i < q; i++) {
        Z[i] = Z[i] / sum_Z;
        temp += Z[i];
    }
    sum_Z = temp;
    printf("Soma das probabilidades: %.20f\nCriando Soma com probabilidade acumulada...\n", sum_Z);

    return Z;
}
__global__ void calculateSums(float *P, float *Soma, double *Z, int r, double q) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < r) {
        double k = (q / r);
        double pos = (idx * k);

        P[2 * idx] = (float)pos;
        Soma[2 * idx] = Z[(int)pos];

        if (idx > 0) {
            Soma[2 * idx] += Soma[2 * (idx - 1)];
        }

        P[2 * idx + 1] = pos + 1;

        if (idx < r - 1) {
            Soma[2 * idx + 1] = Soma[2 * idx];
        } else {
            P[2 * idx + 1] = -1 * (int)(hiprand_uniform(&state) * 101 * q);
            Soma[2 * idx + 1] = 1;
        }
    }
}

__global__ void Soma_P_CUDA(double r, double q, float *P, float *Soma, double *Z, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n) {
        double k = (q / r);
        double pos = tid * k;
        P[2 * tid] = (float)pos;
        atomicAdd(&Soma[2 * tid], Z[(int)pos]);
        P[2 * tid + 1] = pos + 1;
        atomicAdd(&Soma[2 * tid + 1], Z[((int)pos) + 1]);

        if (tid == ((int)r) - 1) {
            P[2 * tid] = -1 * (int)((rand() % 101) * q);
            Soma[2 * tid] = 1;
        }
    }
}

float *Soma_P_CUDA_Wrapper(double r, double q, float *P, float *Soma, double *Z, int n) {
    float *d_P, *d_Soma;
    double *d_Z;

    hipMalloc((void **)&d_P, 2 * n * sizeof(float));
    hipMalloc((void **)&d_Soma, 2 * n * sizeof(float));
    hipMalloc((void **)&d_Z, q * sizeof(double));

    hipMemcpy(d_P, P, 2 * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Soma, Soma, 2 * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Z, Z, q * sizeof(double), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    Soma_P_CUDA<<<blocksPerGrid, threadsPerBlock>>>(r, q, d_P, d_Soma, d_Z, n);

    hipMemcpy(P, d_P, 2 * n * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(Soma, d_Soma, 2 * n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_P);
    hipFree(d_Soma);
    hipFree(d_Z);

    return Soma;
}
int main(){
    double p1 = 31;
    double p2 = 29;
    double N  = p1 * p2; //N nao precisa ser semi-primo
    double x  = 2;
    double r  = 0;
    double q  = (int)pow(2, 24);//2**20
    int n  = 15; // quantidade de valores medidos 
    float *Soma;
    float *P;
    int tamSoma_P;
    float **R;
    float **S;
    int *fat;
    int tamFat;
    double *Z; 
    
    Z = Prepara(N, x, &r, q);
    hipDeviceSynchronize();
    tamSoma_P=(2*(r+1));
    P = (float*)malloc((2*(r+1))*sizeof(float));
    Soma = (float*)malloc((2*r+1)*sizeof(float));
    if (P == NULL || Soma == NULL) {
        printf("Erro na alocacao de memoria.");
        exit(1);
    }
    for(int i=0;i<(2*(r+1));i++){
        P[i]=0;
        Soma[i]=0;
    }
    Soma = Soma_P(r, q, P, Soma, Z);
    
    
    printf("%.0f", r);
    
    return 0;
}