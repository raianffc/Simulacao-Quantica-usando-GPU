#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <stdlib.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#include<stdbool.h>
#include<time.h>
#include<complex.h>

int mdc(int num1, int num2) {
    int resto;
    do {
        resto = num1 % num2;
        num1 = num2;
        num2 = resto;
    } while (resto != 0);
    return num1;
}

int mmc(int num1, int num2) {
    int a;
    if(num2==0) return num1;
    a = mdc(num1,num2);
    return (num1 * num2) / a;
    
}
float buscabin(float *Soma, float *P, double m, int tamSoma) {
    int n = tamSoma;
    if (n == 0) {
        return 0;
    } else if (n == 1) {
        return P[0];
    } else if (n == 2) {
        if (m <= Soma[0]) {
            return P[0];
        } else {
            return P[1];
        }
    } else {
        int meio = n / 2;
        if (m == Soma[meio]) {
            return P[meio];
        } else if (m < Soma[meio]) {
            return buscabin(Soma, P, m, meio);
        } else {
            return buscabin(&Soma[meio], &P[meio], m, n - meio);
        }
    }
}
double* FracCont(double x, double q, double N, int *tamL) {
    int tam = 1;
    double x_inic = x;
    double *L;
    L = (double*)malloc(tam * sizeof(double));
    if (L == NULL) {
        printf("Erro na alocacao de memoria.");
        exit(1);
    }
    if (x == 1) {
        tam = 2;
        L = (double*)realloc(L, tam * sizeof(double));
        if (L == NULL) {
            printf("Erro na alocacao de memoria.");
            exit(1);
        }
        L[0] = 1;
        L[1] = 0;
        x = 0;
    } else {
        L[0] = -1;
    }
    x = x / q;
    int i = 0;
    double max = log(x_inic) / log(1.6);
    do {
        int c = (int) x;
        if (tam != 1) {
            L = (double*)realloc(L, (tam + 1) * sizeof(double));
            if (L == NULL) {
                printf("Erro na alocacao de memoria.");
                exit(1);
            }
            L[tam] = (double) c;
        } else {
            L[0] = (double) c;
        }
        x = x - c;
        if (x >= 0) {
            x = 1 / x;
            if (x > x_inic || x > N) {
                x = 0;
            }
        } else {
            x = 0;
        }
        i++;
        tam++;
    } while (x > 0 && i < max);
    *tamL = tam - 1;
    return L;
}

double** Frac(double *L, int *tamL) {
    int tamF = *tamL;
    if (*tamL == 0) {
        *tamL = 1;
        L = (double*)realloc(L, (*tamL) * sizeof(double));
        if (L == NULL) {
            printf("Erro na alocacao de memoria.");
            exit(1);
        }
        L[0] = 1;
    }
    double **F = (double**)malloc(2 * sizeof(double *));
    if (F == NULL) {
        printf("Erro na alocacao de memoria.");
        exit(1);
    }
    for (int i = 0; i < 2; i++) {
        F[i] = (double*)malloc((tamF) * sizeof(double));
        if (F[i] == NULL) {
            printf("\nErro na alocacao de memoria.\n");
            exit(1);
        }
    }
    for(int i = 0;i<2; i++){
        for(int j = 0; j<(tamF);j++){
            F[i][j]=0.0;
        }
    }
    int num = 1;
    int den = 1;
    for ( int i=0; i<((tamF)-1);i++){
        num = 1;
        den = (int)L[(tamF)-i-1];
        for (int j=((tamF)-i-2); j>0;j--){
            int temp=num;
            num = den;
            den = L[j]*den+temp;
        }
        F[0][i+1]=(double)num;
        F[1][i+1]=(double)den;
    }
    F[0][0]=(tamF)-1;

    int total = (int)F[1][2];
    for(int j=3;j<tamF;j++){
        total = mmc(total,(int)F[1][j]);
    }
    F[1][0]=(double)total;
    if(F[1][0]==0 && *tamL==1){
        printf("\nFracao inexistente: %.0f \n", L[0]);
    }
    free(L);
    return F;
}

__global__ void calculateZ(hipfftComplex *d_Y, int q, int r) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < q) {
        d_Y[idx] = make_hipComplex(0.0f, 0.0f);
    }

    __syncthreads();

    int j = 1;
    while (j <= q) {
        if (idx == j) {
            d_Y[j] = make_hipComplex(1.0f, 0.0f);
        }
        j += r;
        __syncthreads();
    }
}

__global__ void Quadrado_conjugado(double *d_Z, hipfftComplex *d_Y, int q) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < q) {
        d_Z[idx] = hipCabsf(d_Y[idx]) * hipCabsf(d_Y[idx]);
    }
}

__global__ void normalizeZ(double *d_Z, double sum_Z, int q) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < q) {
        d_Z[idx] /= sum_Z;
    }
}
__global__ void sumVector(float *input, int size, float *result) {
    extern __shared__ float sdata[];
    
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = (i < size) ? input[i] : 0.0f;
    
    __syncthreads();
    
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        result[blockIdx.x] = sdata[0];
    }
}


double *Prepara(double N, double x, double *r, double q) {
    int tamN = (int)log2(N);
    double q1 = 1 << (2 * tamN);  
    printf("Valor ideal para q: %.0f\n", q1);

    if (q < N) {
        q = 1 << (tamN + 4);
    }

    if (*r == 0) {
        int s = x;
        int i = 1;
        while (s > 1) {
            s = (int)(s * x) % ((int)N);
            i++;
        }
        *r = i;
        printf("Ordem r nao informada. Ordem r calculada: %.0f\n", *r);
    } else {
        printf("Ordem r informada: %f\n", *r);
    }

    printf("Criando Z...\n");
    double *Z;
    hipHostMalloc((void **)&Z, q * sizeof(double), hipHostMallocDefault);

    // Aloca memória na GPU para Y
    hipfftComplex *d_Y;
    hipMalloc((void **)&d_Y, q * sizeof(hipfftComplex));

    // Define o número de threads por bloco e calcula o número de blocos
    int threadsPerBlock = 256; 
    int numBlocks = (q + threadsPerBlock - 1) / threadsPerBlock;

    // Preenche Y
    calculateZ<<<numBlocks, threadsPerBlock>>>(d_Y, q, (int)(*r));
    hipDeviceSynchronize();

    // Calcula FFT
    printf("Calculando FFT...\n");
    hipfftHandle plan;
    hipfftPlan1d(&plan, q, HIPFFT_C2C, 1);
    hipfftExecC2C(plan, d_Y, d_Y, HIPFFT_FORWARD);
    hipfftDestroy(plan);

    // Calcula probabilidades
    printf("Calculando probabilidades...\n");
    Quadrado_conjugado<<<numBlocks, threadsPerBlock>>>(Z, d_Y, q);
    hipDeviceSynchronize();

    // Calcula a soma das probabilidades
    double sum_Z = 0;
    for (int i = 0; i < q; i++) {
        sum_Z += Z[i];
    }
    double temp=0;
    for (int i = 0; i < q; i++) {
        Z[i] = Z[i]/sum_Z;
        temp += Z[i];
    }
    sum_Z = temp;
    printf("Soma das probabilidades: %.20f\nCriando Soma com probabilidade acumulada...\n", sum_Z);
    


    hipFree(d_Y);
    return Z;
}
/*__global__ void calculateSomaPParallel(float *P, float *Soma, double *Z, double k, double q, int r, curandState *state) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < r) {
        double pos = idx * k;

        P[2 * idx] = (float)pos;
        P[2 * idx + 1] = pos + 1;

        double total = 0.0;
        for (int i = 0; i <= pos; i++) {
            total += Z[i];
        }

        // Gerar um número aleatório entre 0 e 100 usando o cuRAND
        float random_value = curand_uniform(&state[idx]) * 101;

        Soma[2 * idx] = total;
        Soma[2 * idx + 1] = total + Z[(int)pos + 1] * random_value;
    }
}

float *Soma_P_Parallel(double r, double q, float *P, float *Soma, double *Z) {
    double k = (q / r);

    // Aloca estados do cuRAND
    int numThreadsPerBlock = 256;
    int numBlocks = (r + numThreadsPerBlock - 1) / numThreadsPerBlock;
    curandState *devStates;
    cudaMalloc((void **)&devStates, numBlocks * numThreadsPerBlock * sizeof(curandState));
    setupCurand<<<numBlocks, numThreadsPerBlock>>>(devStates, time(0));
    cudaDeviceSynchronize();

    // Configuração dos blocos e threads para o kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (r + threadsPerBlock - 1) / threadsPerBlock;

    // Chama o kernel para calcular Soma_P paralelamente
    calculateSomaPParallel<<<blocksPerGrid, threadsPerBlock>>>(P, Soma, Z, k, q, r, devStates);
    cudaDeviceSynchronize();

    // Libera memória alocada para os estados do cuRAND
    cudaFree(devStates);

    return Soma;
}*/ //Nao sabendo calcular Probabilidade Acumulada (dos picos)
__global__ void generateRandom(float *randomValues, unsigned int seed) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState state;
    hiprand_init(seed, tid, 0, &state);

    randomValues[tid] = hiprand_uniform(&state);
}
float *Soma_P (double r, double q, float *P, float *Soma, double *Z){
    double k = (q / r);
    printf("Calculando Somas...\n");
    double total = 0;

    for (int i = 0; i <r; i++) {
        double pos = (i * k);
        P[2 * i] = (float)pos;
        total += Z[(int)pos];
        Soma[2*i]= total;
        P[2*i+1]= pos+1;
        total = total + Z[((int)pos)+1];
        Soma[2*i+1] = total;
    }
    int numThreads = 1;
    int numBlocks = 1;
    int totalThreads = numThreads * numBlocks;

    float *d_randomValues;
    float *h_randomValues = (float *)malloc(totalThreads * sizeof(float));

    hipMalloc((void **)&d_randomValues, totalThreads * sizeof(float));

    generateRandom<<<numBlocks, numThreads>>>(d_randomValues, time(0));
    
    hipMemcpy(h_randomValues, d_randomValues, totalThreads * sizeof(float), hipMemcpyDeviceToHost);

    P[2*(((int)r)-1)] = -1 * (int)(h_randomValues[0] * 101 * q);
    Soma[2*(((int)r)-1)] = 1;

    printf("Probabilidade Acumulada (dos picos): %f\n", total);
    
    hipFree(d_randomValues);
    free(h_randomValues);

    return Soma;
}
double *Simula(float *Soma,float *P, int tamResult, int tamSoma){
    srand(time(NULL));
    printf("\nSimula medicao:\n");
    double *result;
    result=(double*)malloc(tamResult*sizeof(double));
    if (result == NULL) {
        printf("Erro na alocacao de memoria.");
        exit(1);
    }
    for(int i = 0; i<tamResult; i++) result[i]=0;
    int numThreads = 256; 
    int numBlocks = (tamResult + numThreads - 1) / numThreads;
    int totalThreads = numThreads * numBlocks;

    float *d_randomValues;
    float *h_randomValues = (float *)malloc(totalThreads * sizeof(float));

    hipMalloc((void **)&d_randomValues, totalThreads * sizeof(float));

    generateRandom<<<numBlocks, numThreads>>>(d_randomValues, time(0));
    
    hipMemcpy(h_randomValues, d_randomValues, totalThreads * sizeof(float), hipMemcpyDeviceToHost);

    for(int i = 0; i<tamResult; i++){
    // Normaliza o número para estar entre 0 e 1, incluindo valores decimais
        double m = (double)h_randomValues[i];
        result[i] = (double)buscabin(Soma,P,m, tamSoma);
        if (result[i]==0){
            result[i] = 1;
        }
    }

    hipFree(d_randomValues);
    free(h_randomValues);

    return result;
}
float **EstimaOrdem(double r,double *result,double q, double N, int n){
    float **R;
    int taml = 1;
    R=(float**)malloc(n*sizeof(float*));
    if (R == NULL) {
        printf("Erro na alocacao de memoria.");
        exit(1);
    }
    for(int i=0;i<n;i++){
        R[i]=(float*)malloc(3*sizeof(float));
        if (R[i] == NULL) {
            printf("Erro na alocacao de memoria.");
            exit(1);
        }
    }
    printf("\nTenta estimar a ordem r=ord(x,N) ou multiplo ou divisor dela para extrair os fatores de N\n");
    for (int i=0;i<n;i++){
        for(double j =-1; j<2; j++){
            printf(". ");
            double *l = FracCont(result[i]+j,q,N, &taml);
            double **t = Frac(l, &taml);
            int k = (float)j;
            R[i][k+1]=(float)t[1][0];
            free(t);
        }
    }
    return R;
}
float **EstimaFator(double N, double x,float **R, int tam){
    printf("\nProcura um multiplo da ordem ou um divisor que distiga um fator nao trivial.\n");
    float **Sucesso;
    int potTotal=1;
    int pot;
    Sucesso=(float**)malloc(tam*sizeof(float*));
    if (Sucesso == NULL) {
        printf("Erro na alocacao de memoria.");
        exit(1);
    }
    for(int i = 0; i<tam; i++){
        Sucesso[i]=(float*)malloc(3*sizeof(float));
        if (Sucesso[i] == NULL) {
            printf("Erro na alocacao de memoria.");
            exit(1);
        }
    }
    for(int i = 0; i<tam; i++){
        float sucesso[3]={0,0,0};
        for(int j=0; j<3; j++){
            double x1=(double)x;
            double R1=(double)R[i][j];
            pot=(((int)(pow(x1,R1)))%((int)N));
             if (pot==1){
                sucesso[j]=1;
                printf("\nmultiplo de r\n");
            }
            else{
                int d=mdc(pot-1,N);
                if (d > 1){
                    sucesso[j]=2;
                    printf("\nfator: %d\n",d);
                }
            }
            potTotal = potTotal*pot;
            if(potTotal%((int)N)==1){
                sucesso[j]=sucesso[j]+4;
                printf("\nfatores do múltiplo da ordem %d %d\n",pot,(potTotal/pot));
            }
            else{
                int d=mdc((potTotal-1),N);
                if (d > 1){
                    sucesso[j]=sucesso[j]+8;
                }
            }
            potTotal = potTotal%((int)N);
            //printf("potTotal: %d", potTotal);
            Sucesso[i][j]=sucesso[j];
        }

    }

    return Sucesso;
}
int existe(int valor, int *array, int tamanho) {
    for (int i = 0; i < tamanho; i++) {
        if (array[i] == valor) {
            return 1;  // O valor já existe
        }
    }
    return 0;  // O valor não existe
}
int removerDuplicatas(int *array, int tamanho) {
    if (tamanho <= 1) {
        return tamanho;  // Não há duplicatas para remover
    }

    int novoTamanho = 1;  // Tamanho do novo array sem duplicatas
    for (int i = 1; i < tamanho; i++) {
        if (!existe(array[i], array, novoTamanho)) {
            array[novoTamanho] = array[i];  // Adiciona o elemento único
            novoTamanho++;
        }
    }

    return novoTamanho;
}
int* Fatores(double N, double x, float **R, float **S, int num_s, int *num_fatores) {
    int *fat;
    int *aux;
    fat = (int *)malloc((num_s * num_s * 2) * sizeof(int));
    if (fat == NULL) {
        printf("Erro na alocacao de memoria.");
        exit(1);
    }
    int count = 0;

    for (int i = 0; i < num_s; i++) {
        for (int j = 0; j < 3; j++) {
            if (S[i][j] == 1) {
                if ((int)R[i][j] % 2 == 0) {
                    int f = mdc((int)(((int)pow(x, (int)(R[i][j] / 2))%(int)N) - 1), (int)N);
                    fat[count++] = f;
                }
                if ((int)R[i][j] % 3 == 0) {
                    int f = mdc((int)(((int)pow(x, (int)(R[i][j] / 3))%(int)N) - 1), (int)N);
                    fat[count++] = f;
                }
            } else if (S[i][j] == 2) {
                int f = mdc((int)pow(x, (int)R[i][j]) - 1, (int)N);
                fat[count++] = f;
                fat[count++] = (int)N / f;
            }
        }
    }
   /*int k=1, j=0;
    for(int i=0; i<count;i++){
        if(fat[i]!=N){
            aux = (int*)realloc(aux,k*sizeof(int));
            aux[j] =fat[i];
		    k++;
            j++;
        }
    }*/
    //printf("\ncontador k: %d \n",k);
    /*if(k>1){
        printf("Aux: [");
        for(int i=0; i<k;i++){
            printf("%d ", aux[i]);
        }
        printf("]\n");
    }*/
    *num_fatores = removerDuplicatas(fat, count);
    return fat;
}
int main(){

    //double time_spent = 0.0;
    //clock_t begin = clock();
    double p1 = 31;
    double p2 = 29;
    double N  = p1 * p2; //N nao precisa ser semi-primo
    double x  = 2;
    double r  = 0;
    double q  = (int)pow(2, 24);//2**20
    int n  = 15; // quantidade de valores medidos 
    float *Soma;
    float *P;
    int tamSoma_P;
    float **R;
    float **S;
    int *fat;
    int tamFat;
    double *Z;
    int threadsPerBlock = 256;
    int numBlocks = ( (int)r+ threadsPerBlock - 1) / threadsPerBlock;

    Z = Prepara(N, x, &r, q);

    int r_int = (int)r;

    P = (float *)malloc((2 * (r + 1)) * sizeof(float));
    Soma = (float *)malloc((2 * r + 1) * sizeof(float));
    tamSoma_P=(2*(r+1));
    if (P == NULL || Soma == NULL || Z == NULL) {
        printf("Erro na alocação de memória.");
        exit(1);
    }

    Soma = Soma_P(r, q, P, Soma, Z);

    printf("%.0f", r);

    double *result;
    result = Simula(Soma, P, n, tamSoma_P);
    printf("\nq= %.0f\n",q);
    printf("\nResultados medidos na rotina quantica do QOFA:\n");
    printf("[");
    for(int i=0; i<n;i++){
        printf("%.0f ", result[i]);
    }
    printf("]\n");

    R= EstimaOrdem(r, result, q, N, n);

    S = EstimaFator(N, x, R, n);
    fat = Fatores(N, x, R, S, n, &tamFat);
    printf("\nFatores: \n");
    printf("[");
    for(int i=0; i<tamFat;i++){
        printf("%d ",fat[i]);
    }
    printf("]\n");
    free(fat);
    
    hipHostFree(Z);
    free(P);
    free(Soma);
    //clock_t end = clock();
    //time_spent += (double)(end - begin) / CLOCKS_PER_SEC;
 
    //printf("\nTempo de execucao: %f segundos\n", time_spent);
    return 0;
}