#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <stdlib.h>
#include <math.h>

__global__ void calculateZ(hipfftComplex *d_Y, int q, int r) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < q) {
        d_Y[idx] = make_hipComplex(0.0f, 0.0f);
    }

    __syncthreads();

    int j = 1;
    while (j <= q) {
        if (idx == j) {
            d_Y[j] = make_hipComplex(1.0f, 0.0f);
        }
        j += r;
        __syncthreads();
    }
}

__global__ void calculateProbabilities(double *d_Z, hipfftComplex *d_Y, int q) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < q) {
        d_Z[idx] = hipCabsf(d_Y[idx]) * hipCabsf(d_Y[idx]);
    }
}

__global__ void normalizeZ(double *d_Z, double sum_Z, int q) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < q) {
        d_Z[idx] /= sum_Z;
    }
}
__global__ void reduceSum(double *d_input, double *d_output, int N) {
    extern __shared__ double shared_data[]; // Memória compartilhada

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;

    if (idx < N) {
        shared_data[tid] = d_input[idx];
    } else {
        shared_data[tid] = 0.0;
    }

    __syncthreads();

    // Realiza a redução na memória compartilhada
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_data[tid] += shared_data[tid + s];
        }
        __syncthreads();
    }

    // O thread 0 de cada bloco escreve o resultado final no vetor de saída
    if (tid == 0) {
        d_output[blockIdx.x] = shared_data[0];
    }
}

__global__ void normalizeZ(double *d_Z, double sum_Z, int q) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < q) {
        d_Z[idx] /= sum_Z;
    }
}
__global__ void parallelSoma_P(float *d_P, float *d_Soma, double *d_Z, int r, int q) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < r) {
        double k = (double)q / r;
        double pos = idx * k;

        d_P[2 * idx] = (float)pos;
        double total = 0.0;
        for (int i = 0; i <= idx; i++) {
            total += d_Z[(int)(i * k)];
        }
        d_Soma[2 * idx] = total;
        d_P[2 * idx + 1] = (float)pos + 1;
        d_Soma[2 * idx + 1] = total + d_Z[(int)((idx + 1) * k)];
    }
}

double *Prepara(double N, double x, double *r, double q) {
    int tamN = (int)log2(N);
    double q1 = 1 << (2 * tamN);  
    printf("Valor ideal para q: %.0f\n", q1);

    if (q < N) {
        q = 1 << (tamN + 4);
    }

    if (*r == 0) {
        int s = x;
        int i = 1;
        while (s > 1) {
            s = (int)(s * x) % ((int)N);
            i++;
        }
        *r = i;
        printf("Ordem r não informada. Ordem r calculada: %.0f\n", *r);
    } else {
        printf("Ordem r informada: %f\n", *r);
    }

    printf("Criando Z...\n");
    double *Z;
    hipHostMalloc((void **)&Z, q * sizeof(double));

    // Aloca memória na GPU para Y
    hipfftComplex *d_Y;
    hipMalloc((void **)&d_Y, q * sizeof(hipfftComplex));

    // Define o número de threads por bloco e calcula o número de blocos
    int threadsPerBlock = 256; 
    int numBlocks = (q + threadsPerBlock - 1) / threadsPerBlock;

    // Preenche Y
    calculateZ<<<numBlocks, threadsPerBlock>>>(d_Y, q, (int)(*r));
    hipDeviceSynchronize();

    // Calcula FFT
    printf("Calculando FFT...\n");
    hipfftHandle plan;
    hipfftPlan1d(&plan, q, HIPFFT_C2C, 1);
    hipfftExecC2C(plan, d_Y, d_Y, HIPFFT_FORWARD);
    hipfftDestroy(plan);

    // Calcula probabilidades
    printf("Calculando probabilidades...\n");
    calculateProbabilities<<<numBlocks, threadsPerBlock>>>(Z, d_Y, q);
    hipDeviceSynchronize();

    // Calcula a soma das probabilidades
    double *d_Z;
    double sum_Z;    
    hipMalloc((void **)&d_Z, q * sizeof(double));

    // Copia os dados da CPU para a GPU
    hipMemcpy(d_Z, Z, q * sizeof(double), hipMemcpyHostToDevice);
    int sharedMemorySize = threadsPerBlock * sizeof(double);

    // Executa o kernel de redução para calcular a soma
    reduceSum<<<numBlocks, threadsPerBlock, sharedMemorySize>>>(d_Z, &sum_Z, q);
    hipDeviceSynchronize();

    // Executa o kernel para normalizar Z
    normalizeZ<<<numBlocks, threadsPerBlock>>>(d_Z, sum_Z, q);
    hipDeviceSynchronize();
    //calcula a probabilide entre 0 e 1
    reduceSum<<<numBlocks, threadsPerBlock, sharedMemorySize>>>(d_Z, &sum_Z, q);
    hipDeviceSynchronize();
    // Copia o resultado de volta para a CPU
    hipMemcpy(Z, d_Z, q * sizeof(double), hipMemcpyDeviceToHost);
    printf("Soma das probabilidades: %.20f\nCriando Soma com probabilidade acumulada...\n", sum_Z);


    hipFree(d_Y);
    hipFree(d_Z);
    return Z;
}

float *ParallelSoma_P(double r, double q, float *P, float *Soma, double *Z) {
    int r_int = (int)r;
    int threadsPerBlock = 256;
    int numBlocks = (r_int + threadsPerBlock - 1) / threadsPerBlock;

    float *d_P, *d_Soma;
    double *d_Z;

    hipMalloc((void **)&d_P, (2 * r_int) * sizeof(float));
    hipMalloc((void **)&d_Soma, (2 * r_int) * sizeof(float));
    hipMalloc((void **)&d_Z, q * sizeof(double));
    
    hipMemcpy(d_Z, Z, q * sizeof(double), hipMemcpyHostToDevice);

    parallelSoma_P<<<numBlocks, threadsPerBlock>>>(d_P, d_Soma, d_Z, r_int, (int)q);

    hipMemcpy(P, d_P, (2 * r_int) * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(Soma, d_Soma, (2 * r_int) * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_P);
    hipFree(d_Soma);
    hipFree(d_Z);

    return Soma;
}


int main() {
    double p1 = 31;
    double p2 = 29;
    double N = p1 * p2;
    double x = 2;
    double r = 0;
    double q = pow(2, 20);
    double *Z;
    float *P;
    float *Soma;
    int threadsPerBlock = 256;
    int numBlocks = ( (int)r+ threadsPerBlock - 1) / threadsPerBlock;

    Z = Prepara(N, x, &r, q);

    int r_int = (int)r;
    P = (float *)malloc(2 * r_int * sizeof(float));
    Soma = (float *)malloc(2 * r_int * sizeof(float));
    ParallelSoma_P(r, q, P, Soma, Z);    
    
    
    hipHostFree(Z);
    hipDeviceSynchronize();
    return 0;
}